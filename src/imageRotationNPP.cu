#include "hip/hip_runtime.h"
/* Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
#define WINDOWS_LEAN_AND_MEAN
#define NOMINMAX
#include <windows.h>
#pragma warning(disable : 4819)
#endif


#include <FreeImage.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <cstdio>

// Simple 45°-rotation kernel (nearest-neighbor)
__global__ void rotateKernel(uchar4* in, uchar4* out, int w, int h, float angleRad) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= w || y >= h) return;

    float cx = w * 0.5f, cy = h * 0.5f;
    float xr =  (x - cx) * cosf(angleRad) + (y - cy) * sinf(angleRad) + cx;
    float yr = -(x - cx) * sinf(angleRad) + (y - cy) * cosf(angleRad) + cy;
    int xi = __float2int_rn(xr), yi = __float2int_rn(yr);

    if (xi >= 0 && xi < w && yi >= 0 && yi < h)
        out[y * w + x] = in[yi * w + xi];
    else
        out[y * w + x] = make_uchar4(0,0,0,255);
}

int main(int argc, char** argv) {
    if (argc != 4) {
        printf("Usage: %s input.png output.png angle\n", argv[0]);
        return 1;
    }
    const char* inPath  = argv[1];
    const char* outPath = argv[2];
    float angleDeg = atof(argv[3]);
    float angleRad = angleDeg * 3.14159265f / 180.0f;

    FreeImage_Initialise();
    FIBITMAP* dib = FreeImage_Load(FIF_PNG, inPath);
    FIBITMAP* dib32 = FreeImage_ConvertTo32Bits(dib);
    FreeImage_Unload(dib);
    int w = FreeImage_GetWidth(dib32);
    int h = FreeImage_GetHeight(dib32);
    uchar4* hostData = (uchar4*)FreeImage_GetBits(dib32);

    size_t size = w * h * sizeof(uchar4);
    uchar4 *d_in, *d_out;
    hipMalloc(&d_in,  size);
    hipMalloc(&d_out, size);
    hipMemcpy(d_in, hostData, size, hipMemcpyHostToDevice);

    dim3 block(16,16), grid((w+15)/16,(h+15)/16);
    rotateKernel<<<grid,block>>>(d_in, d_out, w, h, angleRad);
    hipDeviceSynchronize();

    hipMemcpy(hostData, d_out, size, hipMemcpyDeviceToHost);
    hipFree(d_in);
    hipFree(d_out);

    FreeImage_Save(FIF_PNG, dib32, outPath);
    FreeImage_Unload(dib32);
    FreeImage_DeInitialise();
    return 0;
}
